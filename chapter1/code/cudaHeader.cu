#include "hip/hip_runtime.h"
#include"cudaHeader.h"
#include<iostream>

//核函数，计算a+b
__global__ void add(int a,int b,int *c)
{
    //保存a+b的计算结果
    *c=a+b;
}


//cuda测试函数的实现
void cudaTest()
{
    int c;
    //在gpu上开辟一个相同的内存
    int *deviceC;
    hipMalloc((void**)&deviceC,sizeof(int));
    //调用核函数
    add<<<1,1>>>(2,7,deviceC);
    //把计算结果复制到cpu上
    hipMemcpy(&c,deviceC,sizeof(int),hipMemcpyDeviceToHost);
    //展示计算结果
    std::cout<<c<<std::endl;
    //释放内存
    hipFree(deviceC);
}